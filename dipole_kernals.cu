#include "hip/hip_runtime.h"
#include "cuda_objects.cuh"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cmath>
#ifdef KEPLER
#define BLOCK_SIZE 128
#else
#define BLOCK_SIZE 128
#endif

__constant__ intensity_info int_info;


__host__ void copy_intensity_info(intensity_info* int_inf)
{
	//void* ptr;
	//hipGetSymbolAddress( &ptr, HIP_SYMBOL(int_info ));

	hipMemcpyToSymbol(HIP_SYMBOL(int_info), (void*)int_inf, sizeof(intensity_info),0,hipMemcpyHostToDevice);
};

__global__ void device_clear_vector(double* vec,int N){
	int irootI = blockIdx.x * blockDim.x + threadIdx.x;
	if(irootI < N)
		vec[irootI] = 0.0;
}

__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                                          (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, 
                        __double_as_longlong(val + 
                        __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}



__global__ void device_correlate_vectors(cuda_bset_contrT* bset_contr,int idegI,int igammaI,const double* vecI,double* vec)
{
	int irootI = blockIdx.x * blockDim.x + threadIdx.x;
	
	int dimenI = bset_contr->Maxcontracts;
	
	if(irootI < dimenI)
	{
	
		int irow,ib,iterm,nelem,isrootI,Ntot,sdeg;
		double dtemp0 = 0.0;
		irow = bset_contr->icontr2icase[irootI];
		ib = bset_contr->icontr2icase[irootI + bset_contr->Maxcontracts];
	
		iterm = bset_contr->ijterms[irow + igammaI*bset_contr->Maxsymcoeffs];
	
		nelem = bset_contr->N[igammaI + irow*int_info.sym_nrepres];
	
		Ntot = bset_contr->Ntotal[igammaI];
		sdeg = int_info.sym_degen[igammaI];
		double* irr = bset_contr->irr_repres[igammaI];
		
		for(int i = 0; i < nelem; i++)
		{
			isrootI = iterm+i;
			dtemp0 +=  vecI[isrootI]*irr[isrootI + idegI*Ntot + ib*sdeg*Ntot];
		}
	
		vec[irootI] = dtemp0;
	
	}


}
__global__ void device_compute_1st_half_ls(cuda_bset_contrT* bset_contrI,cuda_bset_contrT* bset_contrF,double* dipole_me,int igammaI,double* vector,double* threej,double* half_ls)
{

	const int irootF = blockIdx.x * blockDim.x + threadIdx.x;
	//double sq2 = 1.0/sqrt(2.0);
	
	int dimenI,icontrF, icontrI,kF, kI, tauI,tauF,sigmaF, sigmaI, ktau,dipole_idx,jI,jF;
	//These are o remove if statements
	bool kI_kF_diff,kI_kF_eq,tauF_tauI_neq,kI_kF_zero;
	double ls = 0.0,f3j=0.0,final_half_ls;
	
	dimenI = bset_contrI->Maxcontracts;
	jI = bset_contrI->jval;
	jF = bset_contrF->jval;
	
	if(irootF < bset_contrF->Maxcontracts)
	{
		final_half_ls = 0.0;
	
		//If we are out of range the we always acces the zeroth element
		icontrF = bset_contrF->iroot_correlat_j0[irootF];
	
		tauF  =  bset_contrF->ktau[irootF] & 1;//fmodf((float),2.0f);
		kF = bset_contrF->k[irootF];

		sigmaF = (kF % 3)*tauF;
		//Possible remove this for loop all together
		for(int irootI = 0; irootI < dimenI; irootI++)
		{

		        
		        kI = bset_contrI->k[irootI];
		        kI_kF_diff = fabsf(kI - kF) <= 1.0f;
		        
		        icontrI = bset_contrI->iroot_correlat_j0[irootI];
		        tauI = bset_contrI->ktau[irootI] & 1;
		        
		         sigmaI = (kI % 3)*tauI;
		
			f3j  =  threej[jI + kI*(int_info.jmax+1) + (jF - jI + 1)*(int_info.jmax+1)*(int_info.jmax+1) + (kF - kI +1)*kI_kF_diff*(int_info.jmax+1)*(int_info.jmax+1)*3];  //this is big and unwieldy
		
			
			
		          //Evaluate all conditions without branching
		          kI_kF_eq = (kF==kI); // 1 or 0
		          
		          tauF_tauI_neq = (tauF!=tauI); // 1 or  0
		          
		          kI_kF_zero = ((kI*kF) != 0); // 1 or zero       //If evaluated with CMP branch instruction then implement my own
		          
			  dipole_idx=2*(kI_kF_eq)+(!kI_kF_eq)*(tauF_tauI_neq)*0 + (!kI_kF_eq)*(!tauF_tauI_neq)*1;
			

			  ls = double(tauF-tauI)*double(kI_kF_eq) + (tauF-tauI)*(kF-kI)*(!kI_kF_eq)*( tauF_tauI_neq) + -1.0*(!kI_kF_eq)*(!tauF_tauI_neq);


		          ls*=dipole_me[icontrI + icontrF*int_info.dip_stride_1 + dipole_idx*int_info.dip_stride_2]*(1.0 + (int_info.sq2 - 1.0)*double(kI_kF_zero)*(!kI_kF_eq));
//
		

			  
		          //Only contribue if in range
			// final_half_ls+=pow(-1.0,double(sigmaI+kI))*ls*f3j*vector[irootI]*double(kI_kF_diff);
			final_half_ls+=double(2*((sigmaI+kI) & 1)-1)*ls*f3j*vector[irootI]*double(kI_kF_diff);
			 // half_ls[irootF] +=dipole_idx;//(double)kI_kF_diff;//(double)dipole_idx;
			  
/*
			if(kF==kI)
			   ls = double(tauF-tauI) * dipole_me[icontrI + icontrF*int_info.dip_stride_1 + 2*int_info.dip_stride_2];
			else if(tauF != tauI){
			    ls = double((tauF-tauI)*(kF-kI)) * dipole_me[icontrI + icontrF*int_info.dip_stride_1 + 0*int_info.dip_stride_2];
			    if(kI*kF != 0) ls*=sq2;
			}else if(tauF == tauI){
				ls = -dipole_me[icontrI + icontrF*int_info.dip_stride_1 + 1*int_info.dip_stride_2];
				 if(kI*kF != 0) ls*=sq2;
			}
			half_ls[irootF] +=pow(-1.0,double(sigmaI+kI))*ls*f3j*vector[irootI]*double(kI_kF_diff);
*/				
				

		}
	
		//
		//final_half_ls *= pow(	-1.0	, double(sigmaF) );
		final_half_ls *= double(2*(~(sigmaF) & 1)-1);//pow(	-1.0	, double(sigmaF) );
		half_ls[irootF] = final_half_ls;
	}
		
}

__global__ void device_compute_1st_half_ls_flipped_dipole_safe(const cuda_bset_contrT* bset_contrI,const cuda_bset_contrT* bset_contrF,const double* __restrict__ dipole_me,const double* vector,const double*  __restrict__ threej,double* half_ls)
{

	//const int irootF = blockIdx.x * blockDim.x + threadIdx.x;
	//double sq2 = 1.0/sqrt(2.0);
	const int dimenI = bset_contrI->Maxcontracts;
	int icontrF,icontrI,kF, kI, tauI,tauF,sigmaF, sigmaI, jI,jF,dipole_idx;
	//These are o remove if statements
	int kI_kF_eq,tauF_tauI_neq,kI_kF_zero;
	double ls = 0.0,f3j=0.0,final_half_ls;
	
	
	jI = bset_contrI->jval;
	jF = bset_contrF->jval;
	int* F_iroot_correlat_j0 = bset_contrF->iroot_correlat_j0;

	int* F_k = bset_contrF->k;	
	int* F_ktau =bset_contrF->ktau;

	int* I_iroot_correlat_j0 = bset_contrI->iroot_correlat_j0;
	
	int* I_k = bset_contrI->k;	
	int* I_ktau =bset_contrI->ktau;
	for(int irootF=blockIdx.x * blockDim.x + threadIdx.x; irootF < bset_contrF->Maxcontracts; irootF+=blockDim.x*gridDim.x)
	{
		final_half_ls = 0.0;
	
		//If we are out of range the we always acces the zeroth element
		icontrF = F_iroot_correlat_j0[irootF];
	
		tauF  =  F_ktau[irootF] & 1;
		kF = F_k[irootF];

		sigmaF = (kF % 3)*tauF;


		
		
		for(int irootI=0; irootI < dimenI; irootI++)
		{
			//All non-dipole global accesses
		        kI = I_k[irootI]; 
			icontrI = I_iroot_correlat_j0[irootI];
			tauI = I_ktau[irootI] & 1;
			
		        if(abs(kI-kF) >1) continue;

		        sigmaI = (kI % 3)*tauI;
			sigmaI = 2*(~(sigmaI+kI) & 1)-1;
		
			f3j  =  threej[jI + kI*(int_info.jmax) + (jF - jI + 1)*(int_info.jmax)*(int_info.jmax) + (kF - kI +1)*(int_info.jmax)*(int_info.jmax)*3];  //this is big and unwieldy
			
			//if(fabsf(f3j) < int_info.threshold) continue;
		        //Evaluate all conditions without branching
		        kI_kF_eq = (kF==kI); // 1 or 0
		          
		        tauF_tauI_neq = (tauF!=tauI); // 1 or  0
		          
		        kI_kF_zero = ((kI*kF) != 0); // 1 or zero       //If evaluated with CMP branch instruction then implement my own

		        dipole_idx=2*(kI_kF_eq)+ (!kI_kF_eq)*(!tauF_tauI_neq)*1;
			
			  // These accesses should be coalesed and therefore significantly faster
			ls = dipole_me[icontrF + icontrI*int_info.dip_stride_1 + dipole_idx*int_info.dip_stride_2];

			ls *= (tauF-tauI)*(kI_kF_eq) + (tauF-tauI)*(kF-kI)*(!kI_kF_eq)*( tauF_tauI_neq) - (!kI_kF_eq)*(!tauF_tauI_neq);
			  
		          //Only contribue if in range
			final_half_ls+=double(sigmaI)*ls*f3j*vector[irootI]*(1.0 + (int_info.sq2 - 1.0)*double(kI_kF_zero)*(!kI_kF_eq));
			//icount++;

							
		}
	
		final_half_ls *= double(2*(~(sigmaF) & 1)-1);
		half_ls[irootF] = final_half_ls;
	}
}

__global__ void device_compute_1st_half_ls_flipped_dipole(const cuda_bset_contrT* bset_contrI,const cuda_bset_contrT* bset_contrF,const double* __restrict__ dipole_me,const double* vector,const double*  __restrict__ threej,double* half_ls)
{

	//const int irootF = blockIdx.x * blockDim.x + threadIdx.x;
	//double sq2 = 1.0/sqrt(2.0);
	const int dimenI = bset_contrI->Maxcontracts;
	int icontrF,icontrI,kF, kI, tauI,tauF,sigmaF, sigmaI, jI,jF,dipole_idx;
	//These are o remove if statements
	int kI_kF_eq,tauF_tauI_neq,kI_kF_zero;
	double ls = 0.0,f3j=0.0,final_half_ls;
	
	
	jI = bset_contrI->jval;
	jF = bset_contrF->jval;
	int* F_iroot_correlat_j0 = bset_contrF->iroot_correlat_j0;

	int* F_k = bset_contrF->k;	
	int* F_ktau =bset_contrF->ktau;

	int* I_iroot_correlat_j0 = bset_contrI->iroot_correlat_j0;
	
	int* I_k = bset_contrI->k;	
	int* I_ktau =bset_contrI->ktau;
	for(int irootF=blockIdx.x * blockDim.x + threadIdx.x; irootF < bset_contrF->Maxcontracts; irootF+=blockDim.x*gridDim.x)
	{
		final_half_ls = 0.0;
	
		//If we are out of range the we always acces the zeroth element
		icontrF = F_iroot_correlat_j0[irootF];
	
		tauF  =  F_ktau[irootF] & 1;
		kF = F_k[irootF];

		sigmaF = (kF % 3)*tauF;


		
		
		for(int irootI=0; irootI < dimenI; irootI++)
		{
			//All non-dipole global accesses
		        kI = I_k[irootI]; 
			icontrI = I_iroot_correlat_j0[irootI];
			tauI = I_ktau[irootI] & 1;
			
		        if(abs(kI-kF) >1) continue;

		        sigmaI = (kI % 3)*tauI;
			sigmaI = 2*(~(sigmaI+kI) & 1)-1;
		
			f3j  =  threej[jI + kI*(int_info.jmax) + (jF - jI + 1)*(int_info.jmax)*(int_info.jmax) + (kF - kI +1)*(int_info.jmax)*(int_info.jmax)*3];  //this is big and unwieldy
			
			//if(fabsf(f3j) < int_info.threshold) continue;
		        //Evaluate all conditions without branching
		        kI_kF_eq = (kF==kI); // 1 or 0
		          
		        tauF_tauI_neq = (tauF!=tauI); // 1 or  0
		          
		        kI_kF_zero = ((kI*kF) != 0); // 1 or zero       //If evaluated with CMP branch instruction then implement my own

		        dipole_idx=2*(kI_kF_eq)+ (!kI_kF_eq)*(!tauF_tauI_neq)*1;
			
			  // These accesses should be coalesed and therefore significantly faster
			ls = dipole_me[icontrF + icontrI*int_info.dip_stride_1 + dipole_idx*int_info.dip_stride_2];

			ls *= (tauF-tauI)*(kI_kF_eq) + (tauF-tauI)*(kF-kI)*(!kI_kF_eq)*( tauF_tauI_neq) - (!kI_kF_eq)*(!tauF_tauI_neq);
			  
		          //Only contribue if in range
			final_half_ls+=double(sigmaI)*ls*f3j*vector[irootI]*(1.0 + (int_info.sq2 - 1.0)*double(kI_kF_zero)*(!kI_kF_eq));
			//icount++;

							
		}
	
		final_half_ls *= double(2*(~(sigmaF) & 1)-1);
		half_ls[irootF] = final_half_ls;
	}
}

__global__ void device_compute_1st_half_ls_flipped_dipole_shared(const cuda_bset_contrT* bset_contrI,const cuda_bset_contrT* bset_contrF,int start_idx,const double* __restrict__ dipole_me,const double* vector,const double*  __restrict__ threej,double* half_ls)
{

	//const int irootF = blockIdx.x * blockDim.x + threadIdx.x;
	//double sq2 = 1.0/sqrt(2.0);
	//volatile __shared__ int s_dipole_idx[BLOCK_SIZE];
	volatile __shared__ double s_ls_factor[BLOCK_SIZE];
	volatile __shared__ int s_icontrI[BLOCK_SIZE];
	volatile __shared__ int s_tauI[BLOCK_SIZE];
	volatile __shared__ int s_sigmaI[BLOCK_SIZE];;
	int t_id = threadIdx.x;
	int b_size = 32;//BLOCK_SIZE;
	int b_start = (threadIdx.x/32)*32;
	int w_id = threadIdx.x % 32;
	const int dimenI = bset_contrI->Maxcontracts;
	const int dimenF = bset_contrF->Maxcontracts;
	int icontrF,kF, kI, tauI,tauF,sigmaF, sigmaI, jI,jF,dipole_idx;
	//These are o remove if statements
	int kI_kF_eq,tauF_tauI_neq,kI_kF_zero;
	double ls = 0.0,f3j=0.0,final_half_ls;
	
	jI = bset_contrI->jval;
	jF = bset_contrF->jval;
	int j0dimen = dimenF/((2*jF)+1); //Exploit
	int startF = start_idx*j0dimen;
	int endF = (start_idx+1)*j0dimen;
	int* F_iroot_correlat_j0 = bset_contrF->iroot_correlat_j0;

	int* F_k = bset_contrF->k;	
	int* F_ktau =bset_contrF->ktau;

	int* I_iroot_correlat_j0 = bset_contrI->iroot_correlat_j0;
	
	int* I_k = bset_contrI->k;	
	int* I_ktau =bset_contrI->ktau;
	//s_dipole_idx[t_id]=0;
	s_ls_factor[t_id]=0.0;
	//__syncthreads();
	//for(int irootF=blockIdx.x * blockDim.x + threadIdx.x + startF; irootF < endF; irootF+=blockDim.x*gridDim.x){
	int irootF=blockIdx.x*blockDim.x + threadIdx.x + startF;
	//kF=-10000;
	icontrF=0;
	tauF = 0;

	//for(irootF=blockIdx.x * blockDim.x + threadIdx.x + startF; irootF < endF; irootF+=blockDim.x*gridDim.x){	
		final_half_ls = 0.0;
	if(irootF<endF){
		//If we are out of range the we always acces the zeroth element
		icontrF = F_iroot_correlat_j0[irootF];
		tauF  =  F_ktau[irootF] & 1;
	}
		kF = F_k[startF];

		sigmaF = (kF % 3)*tauF;
		
		for(int b_irootI=0; b_irootI < dimenI; b_irootI+=j0dimen){

			kI = I_k[b_irootI];
			

			if(abs(kI-kF) >1){
				if(kI > kF) 
					break;
				else
					continue;
			}
			f3j  =  threej[jI + kI*(int_info.jmax) + (jF - jI + 1)*(int_info.jmax)*(int_info.jmax) + (kF - kI +1)*(int_info.jmax)*(int_info.jmax)*3];
			//We can simple have one thread read the memory
			for(int w_irootI=b_irootI; w_irootI < b_irootI+j0dimen; w_irootI+=b_size)
			{
				int irootI = w_irootI+w_id;
				//kI=10000000000;
				//All non-dipole global accesses


				//s_dipole_idx[t_id]=0;
				s_ls_factor[t_id]=0.0;
				//s_icontrI[t_id]=0;
			
				//if(abs(kI-kF) <=1){
				if(irootI < b_irootI+j0dimen){
					s_tauI[t_id] = I_ktau[irootI] & 1;
					s_icontrI[t_id] = I_iroot_correlat_j0[irootI];
					

					s_sigmaI[t_id] = (kI % 3);
		
					  //this is big and unwieldy
			
					//if(fabsf(f3j) < int_info.threshold) continue;
					//Evaluate all conditions without branching
					kI_kF_eq = (kF==kI); // 1 or 0
					  
					 // 1 or  0
					  
					kI_kF_zero = ((kI*kF) != 0); // 1 or zero       //If evaluated with CMP branch instruction then implement my own

					  // These accesses should be coalesed and therefore significantly faster
					//ls = (tauF-tauI)*(kI_kF_eq) + (tauF-tauI)*(kF-kI)*(!kI_kF_eq)*( tauF_tauI_neq) - (!kI_kF_eq)*(!tauF_tauI_neq);

					ls =f3j*vector[irootI]*(1.0 + (int_info.sq2 - 1.0)*double(kI_kF_zero)*(!kI_kF_eq));
					s_ls_factor[t_id] = ls;//1.0;//ls;
				}
				//__threadfence();
			
				//__syncthreads();
				for(int i = 0; i < b_size; i++){

					ls= s_ls_factor[i + b_start];

					if(ls==0.0) continue;
					tauI=s_tauI[i + b_start];
					tauF_tauI_neq = (tauF!=tauI);

					dipole_idx=2*(kI_kF_eq)+ (!kI_kF_eq)*(!tauF_tauI_neq);

					ls *= (tauF-tauI)*(kI_kF_eq) + (tauF-tauI)*(kF-kI)*(!kI_kF_eq)*( tauF_tauI_neq) - (!kI_kF_eq)*(!tauF_tauI_neq);

					sigmaI = s_sigmaI[i+b_start]*tauI;
					sigmaI = 2*(~(sigmaI+kI) & 1)-1;
					final_half_ls+= double(sigmaI)*ls*dipole_me[icontrF + s_icontrI[i + b_start]*int_info.dip_stride_1 + dipole_idx*int_info.dip_stride_2];
				} 
				  //Only contribue if in range
				//final_half_ls+=double(sigmaI)*ls*f3j*vector[irootI]*(1.0 + (int_info.sq2 - 1.0)*double(kI_kF_zero)*(!kI_kF_eq));
				//icount++;
			
				//CLEAR
				//s_dipole_idx[t_id]=0;
				s_ls_factor[t_id]=0.0;
				//s_icontrI[t_id]=0;
				//__threadfence();
							
			}
		}
	if(irootF<endF){
		final_half_ls *= double(2*(~(sigmaF) & 1)-1);
		half_ls[irootF] = final_half_ls;
	}
	//	printf("shared: %i %16.8E st: %i s:%i e:%i kF: %i tauF: %i\n",irootF,final_half_ls,start_idx,startF,endF, kF,tauF);

	//}
}
__global__ void device_compute_1st_half_ls_flipped_dipole_shared_blocks(const cuda_bset_contrT* bset_contrI,const cuda_bset_contrT* bset_contrF,const int start_idx,const int startFblock,const int endFblock,const int ncontrF,const double* __restrict__ dipole_me,const double* vector,const double*  __restrict__ threej,double* __restrict__ half_ls)
{

	//const int irootF = blockIdx.x * blockDim.x + threadIdx.x;
	//double sq2 = 1.0/sqrt(2.0);
	volatile __shared__ double s_ls_factor[BLOCK_SIZE];
	volatile __shared__ int s_icontrI[BLOCK_SIZE];
	volatile __shared__ int s_tauI[BLOCK_SIZE];
	volatile __shared__ int s_sigmaI[BLOCK_SIZE];;
	int t_id = threadIdx.x;
	int b_size = 32;//BLOCK_SIZE;
	int b_start = (threadIdx.x/32)*32;
	int w_id = threadIdx.x % 32;
	const int dimenI = bset_contrI->Maxcontracts;
	const int dimenF = bset_contrF->Maxcontracts;
	int icontrF,kF, kI, tauI,tauF,sigmaF, sigmaI, jI,jF,dipole_idx;
	//These are o remove if statements
	int kI_kF_eq,tauF_tauI_neq,kI_kF_zero;
	double ls = 0.0,f3j=0.0,final_half_ls;
	
	jI = bset_contrI->jval;
	jF = bset_contrF->jval;
	int j0dimen = dimenF/((2*jF)+1); //Exploit
	int startF = start_idx*j0dimen;
	int endF = (start_idx+1)*j0dimen;
	int* F_iroot_correlat_j0 = bset_contrF->iroot_correlat_j0;

	int* F_k = bset_contrF->k;	
	int* F_ktau =bset_contrF->ktau;

	int* I_iroot_correlat_j0 = bset_contrI->iroot_correlat_j0;
	
	int* I_k = bset_contrI->k;	
	int* I_ktau =bset_contrI->ktau;

	s_ls_factor[t_id]=0.0;
	int valid = 0;
	//__syncthreads();
	//for(int irootF=blockIdx.x * blockDim.x + threadIdx.x + startF; irootF < endF; irootF+=blockDim.x*gridDim.x){
	int irootF=blockIdx.x*blockDim.x + threadIdx.x + startF;
	//kF=-10000;
	icontrF=-10;
	tauF = 0;

	//for(irootF=blockIdx.x * blockDim.x + threadIdx.x + startF; irootF < endF; irootF+=blockDim.x*gridDim.x){	
		final_half_ls = 0.0;
	if(irootF<endF){
		//If we are out of range the we always acces the zeroth element
		icontrF = F_iroot_correlat_j0[irootF];
		tauF  =  F_ktau[irootF] & 1;
	}

	valid = ((icontrF >=startFblock) && (icontrF < endFblock));

		
		kF = F_k[startF];

		sigmaF = (kF % 3)*tauF;
		
		for(int b_irootI=0; b_irootI < dimenI; b_irootI+=j0dimen){

			kI = I_k[b_irootI];
			

			//Guarenteed no branching
			if(abs(kI-kF) >1){
				if(kI > kF) 
					break;
				else
					continue;
			}
			f3j  =  threej[jI + kI*(int_info.jmax) + (jF - jI + 1)*(int_info.jmax)*(int_info.jmax) + (kF - kI +1)*(int_info.jmax)*(int_info.jmax)*3];
			//We can simple have one thread read the memory
			for(int w_irootI=b_irootI; w_irootI < b_irootI+j0dimen; w_irootI+=b_size)
			{
				int irootI = w_irootI+w_id;
				//kI=10000000000;
				//All non-dipole global accesses
				

				//s_dipole_idx[t_id]=0;
				s_ls_factor[t_id]=0.0;
				//s_icontrI[t_id]=0;
			
				//if(abs(kI-kF) <=1){
				if(irootI < b_irootI+j0dimen){
					s_tauI[t_id] = I_ktau[irootI] & 1;
					s_icontrI[t_id] = I_iroot_correlat_j0[irootI];
					

					s_sigmaI[t_id] = (kI % 3);
		
					  //this is big and unwieldy
			
					//if(fabsf(f3j) < int_info.threshold) continue;
					//Evaluate all conditions without branching
					kI_kF_eq = (kF==kI); // 1 or 0
					  
					 // 1 or  0
					  
					kI_kF_zero = ((kI*kF) != 0); // 1 or zero       //If evaluated with CMP branch instruction then implement my own

					  // These accesses should be coalesed and therefore significantly faster
					//ls = (tauF-tauI)*(kI_kF_eq) + (tauF-tauI)*(kF-kI)*(!kI_kF_eq)*( tauF_tauI_neq) - (!kI_kF_eq)*(!tauF_tauI_neq);

					ls =f3j*vector[irootI]*(1.0 + (int_info.sq2 - 1.0)*double(kI_kF_zero)*(!kI_kF_eq));
					s_ls_factor[t_id] = ls;//1.0;//ls;
				}
				//__threadfence();
				
				//__syncthreads();
				for(int i = 0; i < b_size; i++){

					ls= s_ls_factor[i + b_start];

					if(ls==0.0) continue;
					tauI=s_tauI[i + b_start];
					tauF_tauI_neq = (tauF!=tauI);

					dipole_idx=2*(kI_kF_eq)+ (!kI_kF_eq)*(!tauF_tauI_neq);

					ls *= (tauF-tauI)*(kI_kF_eq) + (tauF-tauI)*(kF-kI)*(!kI_kF_eq)*( tauF_tauI_neq) - (!kI_kF_eq)*(!tauF_tauI_neq);

					sigmaI = s_sigmaI[i+b_start]*tauI;
					sigmaI = 2*(~(sigmaI+kI) & 1)-1;
					if(valid) final_half_ls+= double(sigmaI)*ls*dipole_me[(icontrF-startFblock) + s_icontrI[i + b_start]*ncontrF + dipole_idx*int_info.dip_stride_1*ncontrF];
					
				} 
	
	
				s_ls_factor[t_id]=0.0;
							
			}
		}
	if(irootF<endF && valid){
		final_half_ls *= double(2*(~(sigmaF) & 1)-1);
		half_ls[irootF] = final_half_ls;
	}
}

__global__ void device_compute_1st_half_ls_flipped_dipole_blocks(cuda_bset_contrT* bset_contrI,cuda_bset_contrT* bset_contrF,int startF,int endF,int ncontrF,double* dipole_me,double* vector,double* threej,double* half_ls)
{

	//const int irootF = blockIdx.x * blockDim.x + threadIdx.x;
	//double sq2 = 1.0/sqrt(2.0);
	const int dimenI = bset_contrI->Maxcontracts;
	const int dimenF = bset_contrF->Maxcontracts;
	int irootF,icontrF,icontrI,kF, kI, tauI,tauF,sigmaF, sigmaI, jI,jF,dipole_idx;
	//These are o remove if statements
	bool kI_kF_diff,kI_kF_eq,tauF_tauI_neq,kI_kF_zero;
	double ls = 0.0,f3j=0.0,final_half_ls;
	int* I_iroot_correlat_j0 = bset_contrI->iroot_correlat_j0;
	
	int* I_k = bset_contrI->k;	
	int* I_ktau =bset_contrI->ktau;
	
	jI = bset_contrI->jval;
	jF = bset_contrF->jval;
	
	irootF = blockIdx.x * blockDim.x + threadIdx.x;
	final_half_ls = 0.0;

	if( irootF < dimenF)
	{
			
		//If we are out of range the we always acces the zeroth element
		icontrF = bset_contrF->iroot_correlat_j0[irootF];

		if(icontrF >=startF && icontrF < endF){
			tauF  =  bset_contrF->ktau[irootF] & 1;
			kF = bset_contrF->k[irootF];
			
			sigmaF = (kF % 3)*tauF;

			for(int irootI = 0; irootI < dimenI; irootI++)
			{
				//All non-dipole global accesses
		        	kI = I_k[irootI]; 
				icontrI = I_iroot_correlat_j0[irootI];
				tauI = I_ktau[irootI] & 1;

				//kI_kF_diff = abs(kI-kF) <=1;
				if(abs(kI-kF) > 1) continue;

				sigmaI = (kI % 3)*tauI;
				sigmaI = 2*(~(sigmaI+kI) & 1)-1;
		
				f3j  =  threej[jI + kI*(int_info.jmax) + (jF - jI + 1)*(int_info.jmax)*(int_info.jmax) + (kF - kI +1)*(int_info.jmax)*(int_info.jmax)*3];  //this is big and unwieldy

				//Evaluate all conditions without branching
				kI_kF_eq = (kF==kI); // 1 or 0
				  
				tauF_tauI_neq = (tauF!=tauI); // 1 or  0
				  
				kI_kF_zero = ((kI*kF) != 0); // 1 or zero       //If evaluated with CMP branch instruction then implement my own

				dipole_idx=2*(kI_kF_eq)+(!kI_kF_eq)*(tauF_tauI_neq)*0 + (!kI_kF_eq)*(!tauF_tauI_neq)*1;
				  // These accesses should be coalesed and therefore significantly faster
				

				ls = (tauF-tauI)*(kI_kF_eq) + (tauF-tauI)*(kF-kI)*(!kI_kF_eq)*( tauF_tauI_neq) -((!kI_kF_eq)*(!tauF_tauI_neq));
				ls *= dipole_me[icontrF-startF + icontrI*ncontrF + dipole_idx*ncontrF*int_info.dip_stride_1];  
				  //Only contribue if in range
				final_half_ls+=double(sigmaI)*ls*f3j*vector[irootI]*(1.0 + (int_info.sq2 - 1.0)*double(kI_kF_zero)*(!kI_kF_eq));
							
			}
	
			final_half_ls *= double(2*(~(sigmaF) & 1)-1);
			half_ls[irootF] = final_half_ls;
		}

	}
		
}


__global__ void device_compute_1st_half_ls_flipped_dipole_branch(cuda_bset_contrT* bset_contrI,cuda_bset_contrT* bset_contrF,double* dipole_me,double* vector,double* threej,double* half_ls)
{

	//const int irootF = blockIdx.x * blockDim.x + threadIdx.x;
	//double sq2 = 1.0/sqrt(2.0);
	const int dimenI = bset_contrI->Maxcontracts;
	
	int icontrF,icontrI,kF, kI, tauI,tauF,sigmaF, sigmaI, jI,jF,dipole_idx;
	//These are o remove if statements
	bool kI_kF_diff,kI_kF_eq,tauF_tauI_neq,kI_kF_zero;
	double ls = 0.0,f3j=0.0,final_half_ls;
	
	
	jI = bset_contrI->jval;
	jF = bset_contrF->jval;
	
	
	for(int irootF=blockIdx.x * blockDim.x + threadIdx.x; irootF < bset_contrF->Maxcontracts; irootF+=blockDim.x*gridDim.x)
	{
		final_half_ls = 0.0;
	
		//If we are out of range the we always acces the zeroth element
		icontrF = bset_contrF->iroot_correlat_j0[irootF];
	
		tauF  =  bset_contrF->ktau[irootF] & 1;
		kF = bset_contrF->k[irootF];

		sigmaF = (kF % 3)*tauF;

		for(int irootI = 0; irootI < dimenI; irootI++)
		{
			//All non-dipole global accesses
		        icontrI = bset_contrI->iroot_correlat_j0[irootI];
		        kI = bset_contrI->k[irootI]; 
			tauI = bset_contrI->ktau[irootI] & 1;

		        kI_kF_diff = (((kI - kF)^(kI-kF)>>31) - ( (kI-kF) >> 31 ) ) <= 1;

		        sigmaI = (kI % 3)*tauI;
			sigmaI = 2*(!(sigmaI+kI) & 1)-1;
		
			f3j  =  threej[jI + kI*(int_info.jmax) + (jF - jI + 1)*(int_info.jmax)*(int_info.jmax) + (kF - kI +1)*kI_kF_diff*(int_info.jmax)*(int_info.jmax)*3];  //this is big and unwieldy
			//if(fabsf(f3j) < 0.00000000000000001) continue;
		        //Evaluate all conditions without branching
			if(kF==kI)
			   ls = double(tauF-tauI) * dipole_me[icontrF + icontrI*int_info.dip_stride_1 + 2*int_info.dip_stride_2];
			else if(tauF != tauI){
			    ls = double((tauF-tauI)*(kF-kI)) * dipole_me[icontrF + icontrI*int_info.dip_stride_1 + 0*int_info.dip_stride_2];
			}else if(tauF == tauI){
			    ls = -dipole_me[icontrF + icontrF*int_info.dip_stride_1 + 1*int_info.dip_stride_2];
			}
			if(kF!=kI && kF*kI!=0) ls*=int_info.sq2;
			  
		          //Only contribue if in range
			final_half_ls+=double(sigmaI)*ls*f3j*vector[irootI]*double(kI_kF_diff);	
							
		}
	
		final_half_ls *= double(2*(!(sigmaF) & 1)-1);
		half_ls[irootF] = final_half_ls;
	}
		
}



__global__ void device_compute_1st_half_ls_2D(cuda_bset_contrT* bset_contrI,cuda_bset_contrT* bset_contrF,double* dipole_me,int igammaI,double* vector,double* threej,double* half_ls)
{
	extern __shared__ double s[];
	int irootF = blockIdx.x * blockDim.x + threadIdx.x;
	int irootI = blockIdx.y * blockDim.y + threadIdx.y;
	double result;
	//double sq2 = 1.0/sqrt(2.0);
	
	int dimenI,dimenF,  icontrF, icontrI,kF, kI, tauF, tauI,sigmaF, sigmaI, ktau,dipole_idx,jI,jF;
	//These are o remove if statements
	bool kI_kF_diff,kI_kF_eq,tauF_tauI_neq,kI_kF_zero;
	double ls = 0.0,f3j=0.0;
	
	dimenI = bset_contrI->Maxcontracts;
	dimenF = bset_contrF->Maxcontracts;
	jI = bset_contrI->jval;
	jF = bset_contrF->jval;
	s[blockDim.x * threadIdx.y + threadIdx.x] = 0.0;
	if(irootF < dimenF & irootI < dimenI)
	{
	
	
		//If we are out of range the we always acces the zeroth element
		icontrF = bset_contrF->iroot_correlat_j0[irootF];
	
		ktau = bset_contrF->ktau[irootF];
		tauF  =  fmodf((float)ktau,2.0f);
		kF = bset_contrF->k[irootF];

		        
		kI = bset_contrI->k[irootI];
		kI_kF_diff = fabsf(kI - kF) <= 1.0f;
		        
		icontrI = bset_contrI->iroot_correlat_j0[irootI];
		ktau = bset_contrI->ktau[irootI];
		tauI = fmodf((float)ktau,2.0f);
		        
		sigmaI = fmodf((float)kI, 3.0f)*tauI;

		f3j  =  threej[jI + kI*(int_info.jmax+1) + (jF - jI + 1)*(int_info.jmax+1)*(int_info.jmax+1) + (kF - kI +1)*kI_kF_diff*(int_info.jmax+1)*(int_info.jmax+1)*3];  //this is big and unwieldy
		
			
			
		          //Evaluate all conditions without branching
		kI_kF_eq = (kF==kI); // 1 or 0
		          
		tauF_tauI_neq = (tauF!=tauI); // 1 or  0
		          
		kI_kF_zero = ((kI*kF) != 0); // 1 or zero       //If evaluated with CMP branch instruction then implement my own
		          
		dipole_idx=2*(kI_kF_eq)+(!kI_kF_eq)*(tauF_tauI_neq)*0 + (!kI_kF_eq)*(!tauF_tauI_neq)*1;
			

		ls = double(tauF-tauI)*double(kI_kF_eq) + (tauF-tauI)*(kF-kI)*(!kI_kF_eq)*( tauF_tauI_neq) + -1.0*(!kI_kF_eq)*(!tauF_tauI_neq);


		ls*=dipole_me[icontrI + icontrF*int_info.dip_stride_1 + dipole_idx*int_info.dip_stride_2]*(1.0 + (int_info.sq2 - 1.0)*double(kI_kF_zero)*(!kI_kF_eq));
		s[blockDim.x * threadIdx.y + threadIdx.x] = half_ls[irootF]+=pow(-1.0,double(sigmaI+kI))*ls*f3j*vector[irootI]*double(kI_kF_diff);
		
		__syncthreads();
			//Reduction part//////////////////////////////////Use a better method/////////////////////
		if(threadIdx.x==0 && threadIdx.y==0){	
			for(int y = 0; y < blockDim.y; y++){
				result = 0;
				for(int x = 0; x < blockDim.x; x++)
					result+=s[blockDim.x*y + x];
				atomicAdd(&half_ls[irootF],result);	//Add the result of the block
			}
		}
		
	}


		//
//	half_ls[irootF] *= pow(	-1.0	, double(sigmaF) );
		
}


////////////////////////////////////////////////////////////NEWER VERSION WHERE WE COMPUTE PER ROOTF

__global__ void device_compute_1st_half_ls_dimenF(cuda_bset_contrT* bset_contrI,cuda_bset_contrT* bset_contrF,double* dipole_me,int irootF,int igammaI,int idegI,double* vector,double* threej,double* half_ls)
{
	extern __shared__  double s[];
	int irootI = blockIdx.x * blockDim.x + threadIdx.x;
	s[threadIdx.x]=0.0;
	//double sq2 = 1.0/sqrt(2.0);
	
	int dimenI,dimenF,  icontrF, icontrI,kF, kI, tauF, tauI,sigmaF, sigmaI, ktau,dipole_idx,jI,jF;
	//These are o remove if statements
	bool kI_kF_diff,kI_kF_eq,tauF_tauI_neq,kI_kF_zero;
	double ls = 0.0,f3j=0.0;
	
	dimenI = bset_contrI->Maxcontracts;
	dimenF = bset_contrF->Maxcontracts;
	jI = bset_contrI->jval;
	jF = bset_contrF->jval;
	if(irootI < dimenI){
	
			//If we are out of range the we always acces the zeroth element
		icontrF = bset_contrF->iroot_correlat_j0[irootF];
	
		ktau = bset_contrF->ktau[irootF];
		tauF  =  fmodf((float)ktau,2.0f);
		kF = bset_contrF->k[irootF];

		sigmaF = fmodf((float)kF, 3.0f)*tauF;

				
		kI = bset_contrI->k[irootI];
		kI_kF_diff = fabsf(kI - kF) <= 1.0f;
				
		icontrI = bset_contrI->iroot_correlat_j0[irootI];
		ktau = bset_contrI->ktau[irootI];
		tauI = fmodf((float)ktau,2.0f);
				
		sigmaI = fmodf((float)kI, 3.0f)*tauI;
		
		f3j  =  threej[jI + kI*(int_info.jmax+1) + (jF - jI + 1)*(int_info.jmax+1)*(int_info.jmax+1) + (kF - kI +1)*kI_kF_diff*(int_info.jmax+1)*(int_info.jmax+1)*3];  //this is big and unwieldy
		
			
			
				  //Evaluate all conditions without branching
		kI_kF_eq = (kF==kI); // 1 or 0
				  
		tauF_tauI_neq = (tauF!=tauI); // 1 or  0
				  
		kI_kF_zero = ((kI*kF) != 0); // 1 or zero       //If evaluated with CMP branch instruction then implement my own
				  
		dipole_idx=2*(kI_kF_eq)+(!kI_kF_eq)*(tauF_tauI_neq)*0 + (!kI_kF_eq)*(!tauF_tauI_neq)*1;
			

		ls = double(tauF-tauI)*double(kI_kF_eq) + (tauF-tauI)*(kF-kI)*(!kI_kF_eq)*( tauF_tauI_neq) + -1.0*(!kI_kF_eq)*(!tauF_tauI_neq);


		ls*=dipole_me[icontrI + icontrF*int_info.dip_stride_1 + dipole_idx*int_info.dip_stride_2]*(1.0 + (int_info.sq2 - 1.0)*double(kI_kF_zero)*(!kI_kF_eq));
	//
		

				  
				  //Only contribue if in range
		//half_ls[irootF]+=pow(-1.0,double(sigmaI+kI))*ls*f3j*vector[irootI]*double(kI_kF_diff);
		s[threadIdx.x] = pow(-1.0,double(sigmaI+kI))*ls*f3j*vector[irootI]*double(kI_kF_diff);
	
		__syncthreads();
		//Reduction part//////////////////////////////////Use a better method/////////////////////
		if(threadIdx.x==0){
			for(int i = 1; i < blockDim.x; i++){
				s[threadIdx.x]+=s[i];
			}
			atomicAdd(&half_ls[irootF],s[threadIdx.x]);
		}
	}
		
}

__global__ void device_complete_half_ls_dimenF(cuda_bset_contrT* bset_contrF,double* half_ls){
	int irootF = blockIdx.x * blockDim.x + threadIdx.x;
	int dimenF = bset_contrF->Maxcontracts;
	if(irootF < dimenF){
		int ktau = bset_contrF->ktau[irootF];
		int tauF  =  fmodf((float)ktau,2.0f);
		int kF = bset_contrF->k[irootF];
		int sigmaF = fmodf((float)kF, 3.0f)*tauF;
		half_ls[irootF] *= pow(	-1.0	, double(sigmaF) );
	}
}
